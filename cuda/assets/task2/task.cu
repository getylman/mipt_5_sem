
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>

__global__ void KernelMatrixAdd(int height, int width, int pitch, float *A,
                                float *B, float *result) {
  size_t index = blockDim.x * blockIdx.x + threadIdx.x;
  size_t step = blockDim.x * gridDim.x;
  const size_t len = height * width;
  for (size_t i = index; i < len; i += step) {
    result[i] = A[i] + B[i];
    __syncthreads();
  }
}

void fill(float *x, size_t len) {
  for (size_t i = 0; i < len; ++i) {
    x[i] = 1.0f;
  }
}

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "Invalid num of arguments\n";
    return 1;
  }
  float *d_x, *d_y, *d_result, *h_x, *h_y, *h_result;
  const unsigned height = atoi(argv[1]), width = atoi(argv[2]);
  const size_t len = height * width;
  const unsigned threadPerBlock = atoi(argv[3]);
  const unsigned blockPerGrid = (len + threadPerBlock - 1) / threadPerBlock;
  hipEvent_t start, stop;
  h_x = (float *)malloc(len * sizeof(float));
  h_y = (float *)malloc(len * sizeof(float));
  h_result = (float *)malloc(len * sizeof(float));
  fill(h_x, len);
  fill(h_y, len);
  hipMalloc(&d_x, len * sizeof(float));
  hipMalloc(&d_y, len * sizeof(float));
  hipMalloc(&d_result, len * sizeof(float));
  hipMemcpy(d_x, h_x, len * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, len * sizeof(float), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  KernelMatrixAdd<<<blockPerGrid, threadPerBlock>>>(height, width, 0, d_x, d_y,
                                                    d_result);
  hipEventRecord(stop);

  float milliseconds = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(h_result, d_result, len * sizeof(float), hipMemcpyDeviceToHost);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_result);
  // for (unsigned i = 0; i < height; ++i) {
  //   for (unsigned j = 0; j < width; ++j) {
  //     std::cout << h_result[i * width + j] << ' ';
  //   }
  //   std::cout << '\n';
  // }
  // std::cout << '\n';
  // std::cout << milliseconds << '\n';
  free(h_x);
  free(h_y);
  free(h_result);
  const std::string graph_data = "graph_data2.txt";
  std::ofstream file;
  file.open(graph_data, std::ios::app);
  if (file.is_open()) {
    file << (std::to_string(milliseconds) + "\n").data();
    file.close();
  }
  return 0;
}
