#include "hip/hip_runtime.h"
#include "ScalarMul.cuh"
#define SHAREDSIZE 1024
/*
 * Calculates scalar multiplication for block
 */
__global__ void ScalarMulBlock(int numElements, float *vector1, float *vector2,
                               float *result) {
  const size_t step = gridDim.x * blockDim.x;
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < numElements;
       i += step) {
    result[i] = vector1[i] * vector2[i];
  }
}
