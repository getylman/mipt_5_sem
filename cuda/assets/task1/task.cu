
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>

__global__ void KernelMul(int numElements, float *x, float *y, float *result) {
  size_t index = blockDim.x * blockIdx.x + threadIdx.x;
  size_t step = blockDim.x * gridDim.x;
  for (size_t i = index; i < numElements; i += step) {
    result[i] = x[i] * y[i];
    __syncthreads();
  }
}

void fill(float *x, size_t len) {
  for (size_t i = 0; i < len; ++i) {
    x[i] = 1.0f;
  }
}

int main(int argc, char **argv) {
  if (argc != 3) {
    std::cout << "Invalid num of arguments\n";
    return 1;
  }
  float *d_x, *d_y, *d_result, *h_x, *h_y, *h_result;
  const size_t len = atoi(argv[1]);
  const unsigned threadPerBlock = atoi(argv[2]);
  const unsigned blockPerGrid = (len + threadPerBlock - 1) / threadPerBlock;
  hipEvent_t start, stop;
  h_x = (float *)malloc(len * sizeof(float));
  h_y = (float *)malloc(len * sizeof(float));
  h_result = (float *)malloc(len * sizeof(float));
  fill(h_x, len);
  fill(h_y, len);
  hipMalloc(&d_x, len * sizeof(float));
  hipMalloc(&d_y, len * sizeof(float));
  hipMalloc(&d_result, len * sizeof(float));
  hipMemcpy(d_x, h_x, len * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, len * sizeof(float), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  KernelMul<<<blockPerGrid, threadPerBlock>>>(len, d_x, d_y, d_result);
  hipEventRecord(stop);

  float milliseconds = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(h_result, d_result, len * sizeof(float), hipMemcpyDeviceToHost);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_result);
  // for (unsigned i = 0; i < len; ++i) {
  //   std::cout << h_result[i] << ' ';
  // }
  // std::cout << '\n';
  // std::cout << milliseconds << '\n';
  free(h_x);
  free(h_y);
  free(h_result);
  // return 0;
  const std::string graph_data = "graph_data1.txt";
  std::ofstream file;
  file.open(graph_data, std::ios::app);
  if (file.is_open()) {
    file << (std::to_string(milliseconds) + "\n").data();
    file.close();
  }
  return 0;
}
