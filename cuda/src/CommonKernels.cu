#include "hip/hip_runtime.h"
#include "CommonKernels.cuh"
#define SHAREDSIZE 1024

__global__ void Reduction(float *g_idata, float *g_odata, int numElements) {
  extern __shared__ float sdata[];

  const unsigned loc_id = threadIdx.x;
  const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[loc_id] = (index < numElements) * g_idata[index];
  __syncthreads();

  for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (loc_id < stride) {
      sdata[loc_id] += sdata[loc_id + stride];
    }
    __syncthreads();
  }

  if (loc_id == 0) {
    atomicAdd(g_odata, sdata[0]);
  }
}

__global__ void BlockMultiplication(int numElements, float *vector1,
                                    float *vector2, float *result) {
  extern __shared__ float sdata[];

  const unsigned loc_id = threadIdx.x;
  const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[loc_id] = (index < numElements) * vector1[index] * vector2[index];

  __syncthreads();

  for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (loc_id < stride) {
      sdata[loc_id] += sdata[loc_id + stride];
    }
    __syncthreads();
  }

  if (loc_id == 0) {
    atomicAdd(result, sdata[0]);
  }
}