#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

__global__ void MatrixVectorMul(int height, int width, float *matrix,
                                float *vector, float *result) {
  size_t index = blockDim.x * blockIdx.x + threadIdx.x;
  size_t step = blockDim.x * gridDim.x;
  for (unsigned i = index; i < height; i += step) {
    result[i] = 0;
    for (unsigned j = 0; j < width; ++j) {
      result[i] += matrix[i * width + j] * vector[j];
    }
  }
}
