#include "hip/hip_runtime.h"
#include <CosineVector.cuh>
#include <ScalarMulRunner.cuh>
#include <math.h>
#include <stdio.h>

float CosineVector(int numElements, float *vector1, float *vector2,
                   int blockSize) {
  float length1 = 0.0f;
  float length2 = 0.0f;
  float result = 0.0f;

  length1 = ScalarMulSumPlusReduction(numElements, vector1, vector1, blockSize);
  length2 = ScalarMulSumPlusReduction(numElements, vector2, vector2, blockSize);
  result = ScalarMulSumPlusReduction(numElements, vector1, vector2, blockSize);
  hipDeviceSynchronize();

  float lngtAmulB = sqrt(length1) * sqrt(length2);
  if (lngtAmulB == 0) {
    return 0.0f;
  } else {
    return result / lngtAmulB;
  }
}
