#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "CommonKernels.cuh"
#include "ScalarMul.cuh"
#include "ScalarMulRunner.cuh"

float ScalarMulTwoReductions(int numElements, float *vector1, float *vector2,
                             int blockSize) {
  float *d_x, *d_y, *d_res;
  dim3 block = dim3(blockSize);
  dim3 gridSize = dim3(
      max(1.0, ceil((float)(numElements + blockSize - 1) / (float)blockSize)));
  hipEvent_t start, stop;
  hipMalloc(&d_x, numElements * sizeof(float));
  hipMalloc(&d_y, numElements * sizeof(float));
  hipMalloc(&d_res, sizeof(float));
  hipMemset(d_res, 0, sizeof(float));

  hipMemcpy(d_x, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  BlockMultiplication<<<gridSize, block, (blockSize * sizeof(float))>>>(
      numElements, d_x, d_y, d_res);
  hipDeviceSynchronize();
  hipEventRecord(stop);

  float milliseconds = 0, res = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_res);

  return res;
}

float ScalarMulSumPlusReduction(int numElements, float *vector1, float *vector2,
                                int blockSize) {
  float *d_x, *d_y, *d_res, *d_res_sum, res_sum = 0;
  dim3 block = dim3(blockSize);
  dim3 gridSize = dim3(
      max(1.0, ceil((float)(numElements + blockSize - 1) / (float)blockSize)));
  hipEvent_t start, stop;
  hipMalloc(&d_x, numElements * sizeof(float));
  hipMalloc(&d_y, numElements * sizeof(float));
  hipMalloc(&d_res, numElements * sizeof(float));
  hipMalloc(&d_res_sum, sizeof(float));
  hipMemset(d_res_sum, 0, sizeof(float));

  hipMemcpy(d_x, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  ScalarMulBlock<<<gridSize, block>>>(numElements, d_x, d_y, d_res);

  hipDeviceSynchronize();

  Reduction<<<gridSize, block, (blockSize * sizeof(float))>>>(d_res, d_res_sum,
                                                              numElements);
  hipDeviceSynchronize();

  hipEventRecord(stop);

  float milliseconds = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(&res_sum, d_res_sum, sizeof(float), hipMemcpyDeviceToHost);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_res);
  hipFree(d_res_sum);

  return res_sum;
}