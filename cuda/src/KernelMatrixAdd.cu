#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

__global__ void KernelMatrixAdd(int height, int width, int pitch, float *A,
                                float *B, float *result) {
  size_t index = blockDim.x * blockIdx.x + threadIdx.x;
  size_t jndex = blockDim.y * blockIdx.y + threadIdx.y;
  if (jndex < width && index < height) {
    result[index * pitch + jndex] =
        A[index * pitch + jndex] + B[index * pitch + jndex];
  }
}
