
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>

#define BLOCKSIZE 256

/*
Примечание:
  Желательно самому в начале скомпилисть с разными размерами блока и нахвать все
  объектники соответствующими названиями a32.out, a64.out и тд до 1024 потому
  что размер shared памяти должен быть известен на этапе компиляции.
*/

__global__ void MatrixMul(int heightA, int widthA, int widthB, float *matrixA,
                          float *matrixB, float *matrixResult) {
  unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned loc_index = threadIdx.x;
  for (unsigned i = 0; i < heightA; ++i) {
    for (unsigned j = 0; j < widthB; ++j) {
      matrixResult[i * widthB + j] = 0;
      __shared__ float shared_arr[BLOCKSIZE];
      shared_arr[loc_index] = 0;
      if (tid < widthA) {
        shared_arr[loc_index] =
            matrixA[i * widthA + tid] * matrixB[tid * widthB + j];
      }
      __syncthreads();
      for (unsigned stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (loc_index < stride) {
          shared_arr[loc_index] += shared_arr[loc_index + stride];
        }
        __syncthreads();
      }
      if (loc_index == 0) {
        atomicAdd(&matrixResult[i * widthB + j], shared_arr[0]);
      }
      __syncthreads();
    }
  }
}

void fill(float *x, size_t len) {
  for (size_t i = 0; i < len; ++i) {
    x[i] = 1.0f;
  }
}

void pr(const float *a, unsigned h, unsigned w) {
  for (unsigned i = 0; i < h; ++i) {
    for (unsigned j = 0; j < w; ++j) {
      std::cout << a[i * w + j] << ' ';
    }
    std::cout << '\n';
  }
}

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "Invalid num of arguments\n";
    return 1;
  }
  float *d_x, *d_y, *d_result, *h_x, *h_y, *h_result;
  const unsigned heightA = atoi(argv[1]), widthA = atoi(argv[2]),
                 widthB = atoi(argv[3]);
  const size_t len1 = heightA * widthA, len2 = widthA * widthB,
               len3 = heightA * widthB;
  const unsigned threadPerBlock = BLOCKSIZE;
  const unsigned blockPerGrid = (widthA + threadPerBlock - 1) / threadPerBlock;
  hipEvent_t start, stop;
  h_x = (float *)malloc(len1 * sizeof(float));
  h_y = (float *)malloc(len2 * sizeof(float));
  h_result = (float *)malloc(len3 * sizeof(float));
  fill(h_x, len1);
  fill(h_y, len2);
  hipMalloc(&d_x, len1 * sizeof(float));
  hipMalloc(&d_y, len2 * sizeof(float));
  hipMalloc(&d_result, len3 * sizeof(float));
  hipMemcpy(d_x, h_x, len1 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, len2 * sizeof(float), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  MatrixMul<<<blockPerGrid, threadPerBlock>>>(heightA, widthA, widthB, d_x, d_y,
                                              d_result);
  hipEventRecord(stop);

  float milliseconds = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(h_result, d_result, len3 * sizeof(float), hipMemcpyDeviceToHost);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_result);

  std::cout << "res:\n";
  pr(h_result, heightA, widthB);

  free(h_x);
  free(h_y);
  free(h_result);
  return 0;
  const std::string graph_data = "graph_data6.txt";
  std::ofstream file;
  file.open(graph_data, std::ios::app);
  if (file.is_open()) {
    file << (std::to_string(milliseconds) + "\n").data();
    file.close();
  }
  return 0;
}
