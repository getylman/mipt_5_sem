#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

__global__ void MatrixMul(int heightA, int widthA, int widthB, float *matrixA,
                          float *matrixB, float *matrixResult) {
  const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned jndex = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned stepi = blockDim.x * gridDim.x;
  const unsigned stepj = blockDim.y * gridDim.y;
  float sum = 0;
  for (unsigned i = index; i < heightA; i += stepi) {
    for (unsigned j = jndex; j < widthB; j += stepj) {
      sum = 0;
      for (unsigned k = 0; k < widthA; ++k) {
        sum += matrixA[i * widthA + k] * matrixB[k * widthB + j];
      }
      matrixResult[i * widthB + j] = sum;
    }
  }
}

/*
matrixResult[i * widthB + j] = 0;
      extern __shared__ float shared_arr[];
      shared_arr[loc_index] = 0;
      shared_arr[loc_index] =
          matrixA[i * widthA + tid] * matrixB[tid * widthB + j];
      __syncthreads();
      for (unsigned stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (loc_index < stride) {
          shared_arr[loc_index] += shared_arr[loc_index + stride];
        }
        __syncthreads();
      }
      if (loc_index == 0) {
        atomicAdd(&matrixResult[i * widthB + j], shared_arr[0]);
      }
      __syncthreads();
*//*

for (unsigned i = 0; i < heightA; ++i) {
  for (unsigned j = 0; j < widthB; ++j) {
    matrixResult[i * widthB + j] = 0;
    for (unsigned k = 0; k < widthA; ++k) {
      matrixResult[i * widthB + j] +=
          matrixA[i * widthA + k] * matrixB[k * widthB + j];
    }
  }
}
// work good
*/