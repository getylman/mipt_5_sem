#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define SHAREDSIZE 1024
#define MulTwoReduction '0'

__global__ void Reduction(float *g_idata, float *g_odata, int numElements) {
  extern __shared__ float sdata[];

  unsigned int threadi = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[threadi] = 0.0f;

  if (i < numElements) {
    sdata[threadi] = g_idata[i];
  }

  __syncthreads();

  for (unsigned int j = blockDim.x / 2; j > 0; j >>= 1) {
    if (threadi < j) {
      sdata[threadi] += sdata[threadi + j];
    }
    __syncthreads();
  }

  if (threadi == 0)
    atomicAdd(g_odata, sdata[0]);
}

__global__ void BlockMultiplication(int numElements, float *vector1,
                                    float *vector2, float *result) {
  extern __shared__ float sdata[];

  unsigned int threadi = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[threadi] = 0.0f;

  if (i < numElements) {
    sdata[threadi] = vector1[i] * vector2[i];
  }

  __syncthreads();

  for (unsigned int j = blockDim.x / 2; j > 0; j >>= 1) {
    if (threadi < j) {
      sdata[threadi] += sdata[threadi + j];
    }
    __syncthreads();
  }

  if (threadi == 0)
    atomicAdd(result, sdata[0]);
}

__global__ void ScalarMulBlock(int numElements, float *vector1, float *vector2,
                               float *result) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int gr_i = gridDim.x * blockDim.x;
  for (; i < numElements; i += gr_i) {
    result[i] = vector1[i] * vector2[i];
  }
}

float ScalarMulTwoReductions(int numElements, float *vector1, float *vector2,
                             int blockSize, float *total_time) {
  float *d_x, *d_y, *d_res;
  dim3 block = dim3(blockSize);
  dim3 gridSize = dim3(
      max(1.0, ceil((float)(numElements + blockSize - 1) / (float)blockSize)));
  hipEvent_t start, stop;
  hipMalloc(&d_x, numElements * sizeof(float));
  hipMalloc(&d_y, numElements * sizeof(float));
  hipMalloc(&d_res, sizeof(float));
  hipMemset(d_res, 0, sizeof(float));

  hipMemcpy(d_x, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  BlockMultiplication<<<gridSize, block, (SHAREDSIZE * sizeof(float))>>>(
      numElements, d_x, d_y, d_res);
  hipDeviceSynchronize();
  hipEventRecord(stop);

  float milliseconds = 0, res = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_res);
  *total_time += milliseconds;

  return res;
}

float ScalarMulSumPlusReduction(int numElements, float *vector1, float *vector2,
                                int blockSize, float *total_time) {
  float *d_x, *d_y, *d_res, *d_res_sum, res_sum = 0;
  dim3 block = dim3(blockSize);
  dim3 gridSize = dim3(
      max(1.0, ceil((float)(numElements + blockSize - 1) / (float)blockSize)));
  hipEvent_t start, stop;
  hipMalloc(&d_x, numElements * sizeof(float));
  hipMalloc(&d_y, numElements * sizeof(float));
  hipMalloc(&d_res, numElements * sizeof(float));
  hipMalloc(&d_res_sum, sizeof(float));
  hipMemset(d_res_sum, 0, sizeof(float));

  hipMemcpy(d_x, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  ScalarMulBlock<<<gridSize, block>>>(numElements, d_x, d_y, d_res);

  hipDeviceSynchronize();

  Reduction<<<gridSize, block, (SHAREDSIZE * sizeof(float))>>>(d_res, d_res_sum,
                                                               numElements);
  hipDeviceSynchronize();

  hipEventRecord(stop);

  float milliseconds = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(&res_sum, d_res_sum, sizeof(float), hipMemcpyDeviceToHost);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_res);
  hipFree(d_res_sum);
  *total_time += milliseconds;

  return res_sum;
}

float CosineVector2(int numElements, float *vector1, float *vector2,
                    int blockSize) {
  float length1 = 0.0f;
  float length2 = 0.0f;
  float result = 0.0f;
  float total_time = 0.0f;

  length1 = ScalarMulSumPlusReduction(numElements, vector1, vector1, blockSize,
                                      &total_time);
  length2 = ScalarMulSumPlusReduction(numElements, vector2, vector2, blockSize,
                                      &total_time);
  result = ScalarMulSumPlusReduction(numElements, vector1, vector2, blockSize,
                                     &total_time);
  hipDeviceSynchronize();

  const std::string graph_data = "graph_data52.txt";
  std::ofstream file;
  file.open(graph_data, std::ios::app);
  if (file.is_open()) {
    file << (std::to_string(total_time) + "\n").data();
    file.close();
  }

  float lngtAmulB = sqrt(length1) * sqrt(length2);
  // printf("%f\n", lngtAmulB);
  if (lngtAmulB == 0) {
    return 0.0f;
  } else {
    return result / lngtAmulB;
  }
}

float CosineVector1(int numElements, float *vector1, float *vector2,
                    int blockSize) {
  float length1 = 0.0f;
  float length2 = 0.0f;
  float result = 0.0f;
  float total_time = 0.0f;

  length1 = ScalarMulTwoReductions(numElements, vector1, vector1, blockSize,
                                   &total_time);
  length2 = ScalarMulTwoReductions(numElements, vector2, vector2, blockSize,
                                   &total_time);
  result = ScalarMulTwoReductions(numElements, vector1, vector2, blockSize,
                                  &total_time);
  hipDeviceSynchronize();

  const std::string graph_data = "graph_data51.txt";
  std::ofstream file;
  file.open(graph_data, std::ios::app);
  if (file.is_open()) {
    file << (std::to_string(total_time) + "\n").data();
    file.close();
  }

  float lngtAmulB = sqrt(length1) * sqrt(length2);
  // printf("%f\n", lngtAmulB);
  if (lngtAmulB == 0) {
    return 0.0f;
  } else {
    return result / lngtAmulB;
  }
}

void fill(float *a, unsigned len) {
  for (unsigned i = 0; i < len; ++i) {
    a[i] = 1.0f;
  }
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("invalid number of arguments\n");
    return 1;
  }
  const size_t kLen = atoi(argv[1]);
  const size_t kBlockSize = atoi(argv[2]);
  float *A, *B;
  A = (float *)malloc(kLen * sizeof(float));
  B = (float *)malloc(kLen * sizeof(float));
  fill(A, kLen);
  fill(B, kLen);
  float res = 0;
  if (argv[3][0] == MulTwoReduction) {
    res = CosineVector1(kLen, A, B, kBlockSize);
  } else {
    res = CosineVector2(kLen, A, B, kBlockSize);
  }
  free(A);
  free(B);
}