
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>

__global__ void MatrixVectorMul(int height, int width, float *matrix,
                                float *vector, float *result) {
  size_t index = blockDim.x * blockIdx.x + threadIdx.x;
  size_t step = blockDim.x * gridDim.x;
  for (unsigned i = index; i < height; i += step) {
    result[i] = 0;
    for (unsigned j = 0; j < width; ++j) {
      result[i] += matrix[i * width + j] * vector[j];
    }
  }
}

void fill(float *x, size_t len) {
  for (size_t i = 0; i < len; ++i) {
    x[i] = 1.0f;
  }
}

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "Invalid num of arguments\n";
    return 1;
  }
  float *d_x, *d_y, *d_result, *h_x, *h_y, *h_result;
  const unsigned height = atoi(argv[1]), width = atoi(argv[2]);
  const size_t len = height * width;
  const unsigned threadPerBlock = atoi(argv[3]);
  const unsigned blockPerGrid = (len + threadPerBlock - 1) / threadPerBlock;
  hipEvent_t start, stop;
  h_x = (float *)malloc(len * sizeof(float));
  h_y = (float *)malloc(width * sizeof(float));
  h_result = (float *)malloc(height * sizeof(float));
  fill(h_x, len);
  fill(h_y, width);
  hipMalloc(&d_x, len * sizeof(float));
  hipMalloc(&d_y, width * sizeof(float));
  hipMalloc(&d_result, height * sizeof(float));
  hipMemcpy(d_x, h_x, len * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, width * sizeof(float), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  MatrixVectorMul<<<blockPerGrid, threadPerBlock>>>(height, width, d_x, d_y,
                                                    d_result);
  hipEventRecord(stop);

  float milliseconds = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(h_result, d_result, height * sizeof(float),
             hipMemcpyDeviceToHost);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_result);
  // for (unsigned i = 0; i < width; ++i) {
  //   // for (unsigned j = 0; j < width; ++j) {
  //   std::cout << h_result[i] << ' ';
  //   // }
  //   std::cout << '\n';
  // }
  // std::cout << '\n';
  // std::cout << milliseconds << '\n';
  free(h_x);
  free(h_y);
  free(h_result);
  const std::string graph_data = "graph_data3.txt";
  std::ofstream file;
  file.open(graph_data, std::ios::app);
  if (file.is_open()) {
    file << (std::to_string(milliseconds) + "\n").data();
    file.close();
  }
  return 0;
}
