#include "hip/hip_runtime.h"
#include <KernelMul.cuh>

__global__ void KernelMul(int numElements, float *x, float *y, float *result) {
  size_t index = blockDim.x * blockIdx.x + threadIdx.x;
  size_t step = blockDim.x * gridDim.x;
  for (size_t i = index; i < numElements; i += step) {
    result[i] = x[i] * y[i];
  }
}
